#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
int main(int argc, char **argv) {
	printf("%s Starting...\n", argv[0]);
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);
	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n",(int)error_id, hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	if (deviceCount == 0) {
		printf("There are no available device(s) that support CUDA\n");
	} else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;
	dev =0;
	hipSetDevice(dev);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);
	printf("Device %d: \"%s\"\n", dev, deviceProp.name);
	hipDriverGetVersion(&driverVersion);
	hipRuntimeGetVersion(&runtimeVersion);
	printf(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n",
		driverVersion/1000, (driverVersion%100)/10,
		runtimeVersion/1000, (runtimeVersion%100)/10);

	printf(" CUDA Capability Major/Minor version number: %d.%d\n",
		deviceProp.major, deviceProp.minor);
	printf(" Total amount of global memory: %.2f MBytes (%llu bytes)\n",
		(float)deviceProp.totalGlobalMem/(pow(1024.0,3)),
		(unsigned long long) deviceProp.totalGlobalMem);
	printf(" GPU Clock rate: %.0f MHz (%0.2f GHz)\n",
		deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
	printf(" Memory Clock rate: %.0f Mhz\n",
		deviceProp.memoryClockRate * 1e-3f);
	printf(" Memory Bus Width: %d-bit\n",
		deviceProp.memoryBusWidth);
	if (deviceProp.l2CacheSize) {
		printf(" L2 Cache Size: %d bytes\n",
		deviceProp.l2CacheSize);
	}
	printf(" Max Texture Dimension Size (x,y,z) 1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
		deviceProp.maxTexture1D
		, deviceProp.maxTexture2D[0],
		deviceProp.maxTexture2D[1],
		deviceProp.maxTexture3D[0], deviceProp.maxTexture3D[1],
		deviceProp.maxTexture3D[2]);
	printf(" Max Layered Texture Size (dim) x layers 1D=(%d) x %d, 2D=(%d,%d) x %d\n",
		deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
		deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
		deviceProp.maxTexture2DLayered[2]);
	printf(" Total amount of constant memory: deviceProp.totalConstMem);
	printf(" Total amount of shared memory per block: "deviceProp.sharedMemPerBlock);
	printf(" Total number of registers available per block:
	deviceProp.regsPerBlock);
	printf(" Warp size:
	printf(" Maximum number of threads per multiprocessor:
	deviceProp.maxThreadsPerMultiProcessor);
	printf(" Maximum number of threads per block:
	deviceProp.maxThreadsPerBlock);
	printf(" Maximum sizes of each dimension of a block:
	deviceProp.maxThreadsDim[0],
	deviceProp.maxThreadsDim[1],
	deviceProp.maxThreadsDim[2]);
	printf(" Maximum sizes of each dimension of a grid:
	deviceProp.maxGridSize[0],
	deviceProp.maxGridSize[1],
	deviceProp.maxGridSize[2]);
	printf(" Maximum memory pitch:
	memPitch);
	❘ 63
	%lu bytes\n",
	%lu bytes\n",
	%d\n",
	%d\n", deviceProp.warpSize);
	%d\n",
	%d\n",
	%d x %d x %d\n",
	%d x %d x %d\n",
	%lu bytes\n", deviceProp.
	exit(EXIT_SUCCESS);
}